#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"


// return GB/sec
float GBPerSec(int bytes, float sec) {
  return static_cast<float>(bytes) / (1024. * 1024. * 1024.) / sec;
}


// This is the CUDA "kernel" function that is run on the GPU.  You
// know this because it is marked as a __global__ function.
__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {

    // compute overall thread index from position of thread in current
    // block, and given the block we are in (in this example only a 1D
    // calculation is needed so the code only looks at the .x terms of
    // blockDim and threadIdx.
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // this check is necessary to make the code work for values of N
    // that are not a multiple of the thread block size (blockDim.x)
    if (index < N)
       result[index] = alpha * x[index] + y[index];
    
    return;
}


// saxpyCuda --
//
// This function is regular C code running on the CPU.  It allocates
// memory on the GPU using CUDA API functions, uses CUDA API functions
// to transfer data from the CPU's memory address space to GPU memory
// address space, and launches the CUDA kernel function on the GPU.
void saxpyCuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {

    // must read both input arrays (xarray and yarray) and write to
    // output array (resultarray)
    int totalBytes = sizeof(float) * 3 * N;

    // compute number of blocks and threads per block.  In this
    // application we've hardcoded thread blocks to contain 512 CUDA
    // threads.
    const int threadsPerBlock = 512;

    // Notice the round up here.  The code needs to compute the number
    // of threads blocks needed such that there is one thread per
    // element of the arrays.  This code is written to work for values
    // of N that are not multiples of threadPerBlock.
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    // These are pointers that will be pointers to memory allocated
    // *one the GPU*.  You should allocate these pointers via
    // hipMalloc.  You can access the resulting buffers from CUDA
    // device kernel code (see the kernel function saxpy_kernel()
    // above) but you cannot access the contents these buffers from
    // this thread. CPU threads cannot issue loads and stores from GPU
    // memory!
    float* device_x = nullptr;
    float* device_y = nullptr;
    float* device_result = nullptr;
    hipMalloc(&device_x, totalBytes);
    hipMalloc(&device_y, totalBytes);
    hipMalloc(&device_result, totalBytes);
    //
    // CS149 TODO: allocate device memory buffers on the GPU using hipMalloc.
    //
    // We highly recommend taking a look at NVIDIA's
    // tutorial, which clearly walks you through the few lines of code
    // you need to write for this part of the assignment:
    //
    // https://devblogs.nvidia.com/easy-introduction-cuda-c-and-c/
    //
        
    // start timing after allocation of device memory
    
    double startTime = CycleTimer::currentSeconds();
    hipMemcpy(device_x, xarray, N, hipMemcpyHostToDevice);
    hipMemcpy(device_y, yarray, N, hipMemcpyHostToDevice);
    //
    // CS149 TODO: copy input arrays to the GPU using hipMemcpy
    //
    
   
    // run CUDA kernel. (notice the <<< >>> brackets indicating a CUDA
    // kernel launch) Execution on the GPU occurs here.
    double KernelTimeStart = CycleTimer::currentSeconds();
    saxpy_kernel<<<blocks, threadsPerBlock>>>(N, alpha, device_x, device_y, device_result);
    hipDeviceSynchronize();
    double KernelTimeEnd = CycleTimer::currentSeconds();
    //
    // CS149 TODO: copy result from GPU back to CPU using hipMemcpy
    //
    hipMemcpy(resultarray, device_result, N, hipMemcpyDeviceToHost);
    
    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n",
		errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    double KernelDuration = KernelTimeEnd - KernelTimeStart;
    printf("Effective BW by CUDA saxpy: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, GBPerSec(totalBytes, overallDuration));
    printf("Effective Kernel by CUDA saxpy: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * KernelDuration, GBPerSec(totalBytes, KernelDuration));
    //
    // CS149 TODO: free memory buffers on the GPU using hipFree
    //
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_result);
    return;
    
}

void printCudaInfo() {

    // print out stats about the GPU in the machine.  Useful if
    // students want to know what GPU they are running on.

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
